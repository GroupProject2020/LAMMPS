#include "hip/hip_runtime.h"
#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float4> vel_tex;
texture<float> cv_tex;
texture<float> e_tex;
texture<float> rho_tex;
texture<float> de_tex;
texture<float> drho_tex;
#else
texture<int4,1> pos_tex;
texture<int4,1> vel_tex;
texture<int2> cv_tex;
texture<int2> e_tex;
texture<int2> rho_tex;
texture<int2> de_tex;
texture<int2> drho_tex;
#endif
#else
#define pos_tex x_
#define vel_tex v_
#define cv_tex cv_
#define e_tex e_
#define rho_tex rho_
#define de_tex de_
#define drho_tex drho_
#endif

__kernel void k_lj_sph(const __global numtyp4 *restrict x_,
                       const __global numtyp4 *restrict v_,
                       const __global numtyp *restrict cv_,
                       const __global numtyp *restrict e_,
                       const __global numtyp *restrict rho_,
                       const __global numtyp *restrict de_,
                       const __global numtyp *restrict drho_,
                       const __global numtyp4 *restrict cuts,
                       const int lj_types,
                       const __global int * dev_nbor,
                       const __global int * dev_packed,
                       __global acctyp4 *restrict ans,
                       __global acctyp *restrict engv,
                       const int eflag, const int vflag, const int inum,
                       const int nbor_pitch,
                       const int t_per_atom){ //TODO: arguments?
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  double h, ih, ihsq, ihcub, wfd, fi, ci, fj, cj;
  double delVdotDelR, mu, fvisc, fpair, deltaE, imass, jmass;

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    double cvi; fetch(cvi, i, cv_tex);
    double ei; fetch(ei, i, cv_tex);
    double rhoi; fetch(rhoi, i, cv_tex);

    LJEOS2(rhoi,ei,cvi, &fi; &ci);
    fi /= (rhoi * rhoi);

    int itype=ix.w;
    imass = cuts[itype].z;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtype=jx.w;


      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;
      jmass = cuts[jtype].z;

      int mtype=itype*lj_types+jtype;
      if (rsq<cuts[mtype].z) {
        h = cuts[mtype].y;
        ih = 1.0/h;
        ihsq = ih * ih;
        ihcub = ihsq * ih;

        wfd = h - sqrt(rsq);

      if (domainDim == 3) {
          // Lucy Kernel, 3d
          // Note that wfd, the derivative of the weight function with respect to r,
          // is lacking a factor of r.
          // The missing factor of r is recovered by
          // (1) using delV . delX instead of delV . (delX/r) and
          // (2) using f[i][0] += delx * fpair instead of f[i][0] += (delx/r) * fpair
          wfd = -25.066903536973515383e0 * wfd * wfd * ihsq * ihsq * ihsq * ih;
        } else {
            // Lucy Kernel, 2d
            wfd = -19.098593171027440292e0 * wfd * wfd * ihsq * ihsq * ihsq;
        }

        double cvj; fetch(cvj, j, cv_tex);
        double ej; fetch(ej, j, cv_tex);
        double rhoj; fetch(rhoj, j, cv_tex);

        // function call to LJ EOS
        LJEOS2(rhoj, ej, cvj, &fj, &cj);
        fj /= (rhoj * rhoj);

        // apply long-range correction to model a LJ fluid with cutoff
        // this implies that the modelled LJ fluid has cutoff == SPH cutoff
        lrc = - 11.1701 * (ihcub * ihcub * ihcub - 1.5 * ihcub);
        fi += lrc;
        fj += lrc;


        // dot product of velocity delta and distance vector
        delVdotDelR = delx * (iv.x - jv.x) + dely * (iv.y -jv.y)
            + delz * (iv.z - jv.z);

        // artificial viscosity (Monaghan 1992)
        if (delVdotDelR < 0.) {
          mu = h * delVdotDelR / (rsq + 0.01 * h * h);
          fvisc = -0.04 * (ci + cj) * mu / (rhoi + rhoj); //TODO: implement the viscosity
        } else {
          fvisc = 0.;
        }

        // total pair force & thermal energy increment
        fpair = -imass * jmass * (fi + fj + fvisc) * wfd;
        deltaE = -0.5 * fpair * delVdotDelR;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        double dei; fetch(dei, i, dcv_tex);
        double drhoi; fetch(drhoi, i, dcv_tex);

        // and change in density
        drhoi += jmass * delVdotDelR * wfd;

        // change in thermal energy
        dei += deltaE;

        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor TODO: write an adapted store_answer function
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

__kernel void k_lj_sph_fast(const __global numtyp4 *restrict x_,
                       const __global numtyp4 *restrict v_,
                       const __global numtyp *restrict cv_,
                       const __global numtyp *restrict e_,
                       const __global numtyp *restrict rho_,
                       const __global numtyp *restrict de_,
                       const __global numtyp *restrict drho_,
                       const __global numtyp4 *restrict cuts_in,
                       const __global int * dev_nbor,
                       const __global int * dev_packed,
                       __global acctyp4 *restrict ans,
                       __global acctyp *restrict engv,
                       const int eflag, const int vflag, const int inum,
                       const int nbor_pitch,
                       const int t_per_atom){ //TODO: arguments?
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  double h, ih, ihsq, ihcub, wfd, fi, ci, fj, cj;
  double delVdotDelR, mu, fvisc, fpair, deltaE, imass, jmass;

  __local numtyp4 cuts[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    cuts[tid]=cuts_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    double cvi; fetch(cvi, i, cv_tex);
    double ei; fetch(ei, i, cv_tex);
    double rhoi; fetch(rhoi, i, cv_tex);

    LJEOS2(rhoi,ei,cvi, &fi; &ci);
    fi /= (rhoi * rhoi);
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    imass = cuts[itype].z;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtype=jx.w;


      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;
      jmass = cuts[jtype].z;

      int mtype=itype+jtype;
      if (rsq<cuts[mtype].z) {
        h = cuts[mtype].y;
        ih = 1.0/h;
        ihsq = ih * ih;
        ihcub = ihsq * ih;

        wfd = h - sqrt(rsq);

      if (domainDim == 3) {
          // Lucy Kernel, 3d
          // Note that wfd, the derivative of the weight function with respect to r,
          // is lacking a factor of r.
          // The missing factor of r is recovered by
          // (1) using delV . delX instead of delV . (delX/r) and
          // (2) using f[i][0] += delx * fpair instead of f[i][0] += (delx/r) * fpair
          wfd = -25.066903536973515383e0 * wfd * wfd * ihsq * ihsq * ihsq * ih;
        } else {
            // Lucy Kernel, 2d
            wfd = -19.098593171027440292e0 * wfd * wfd * ihsq * ihsq * ihsq;
        }

        double cvj; fetch(cvj, j, cv_tex);
        double ej; fetch(ej, j, cv_tex);
        double rhoj; fetch(rhoj, j, cv_tex);

        // function call to LJ EOS
        LJEOS2(rhoj, ej, cvj, &fj, &cj);
        fj /= (rhoj * rhoj);

        // apply long-range correction to model a LJ fluid with cutoff
        // this implies that the modelled LJ fluid has cutoff == SPH cutoff
        lrc = - 11.1701 * (ihcub * ihcub * ihcub - 1.5 * ihcub);
        fi += lrc;
        fj += lrc;


        // dot product of velocity delta and distance vector
        delVdotDelR = delx * (iv.x - jv.x) + dely * (iv.y -jv.y)
            + delz * (iv.z - jv.z);

        // artificial viscosity (Monaghan 1992)
        if (delVdotDelR < 0.) {
          mu = h * delVdotDelR / (rsq + 0.01 * h * h);
          fvisc = -0.04 * (ci + cj) * mu / (rhoi + rhoj); //TODO: implement the viscosity
        } else {
          fvisc = 0.;
        }

        // total pair force & thermal energy increment
        fpair = -imass * jmass * (fi + fj + fvisc) * wfd;
        deltaE = -0.5 * fpair * delVdotDelR;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        double dei; //fetch(dei, i, dcv_tex);
        double drhoi; //fetch(drhoi, i, dcv_tex);

        // and change in density
        drho_[i] += jmass * delVdotDelR * wfd;

        // change in thermal energy
        de_[i] += deltaE;

        if (vflag>0) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor TODO: write an adapted store_answer function
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}
