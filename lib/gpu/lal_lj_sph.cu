#include "hip/hip_runtime.h"
#ifdef NV_KERNEL
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
texture<float4> pos_tex;
texture<float4> vel_tex;
texture<float> cv_tex;
texture<float> e_tex;
texture<float> rho_tex;
texture<float> de_tex;
texture<float> drho_tex;
#else
texture<int4,1> pos_tex;
texture<int4,1> vel_tex;
texture<int2> cv_tex;
texture<int2> e_tex;
texture<int2> rho_tex;
texture<int2> de_tex;
texture<int2> drho_tex;
#endif
#else
#define pos_tex x_
#define vel_tex v_
#define cv_tex cv_
#define e_tex e_
#define rho_tex rho_
#define de_tex de_
#define drho_tex drho_
#endif

__device__ void LJEOS2(double rho, double e, double cv, double *p, double *c) {
  double T = e/cv;
  double beta = 1.0 / T;
  double beta_sqrt = sqrt(beta);
  double x = rho * sqrt(beta_sqrt);

  double xsq = x * x;
  double xpow3 = xsq * x;
  double xpow4 = xsq * xsq;

  /* differential of Helmholtz free energy w.r.t. x */
  double diff_A_NkT = 3.629 + 7.264*x - beta*(3.492 - 18.698*x + 35.505*xsq - 31.816*xpow3 + 11.195*xpow4)
                    - beta_sqrt*(5.369 + 13.16*x + 18.525*xsq - 17.076*xpow3 + 9.32*xpow4)
                    + 10.4925*xsq + 11.46*xpow3 + 2.176*xpow4*xpow4*x;

 /* differential of Helmholtz free energy w.r.t. x^2 */
  double d2A_dx2 = 7.264 + 20.985*x \
                 + beta*(18.698 - 71.01*x + 95.448*xsq - 44.78*xpow3)\
                 - beta_sqrt*(13.16 + 37.05*x - 51.228*xsq + 37.28*xpow3)\
                 + 34.38*xsq + 19.584*xpow4*xpow4;

  // p = rho k T * (1 + rho * d(A/(NkT))/drho)
  // dx/drho = rho/x
  *p = rho * T * (1.0 + diff_A_NkT * x); // pressure
  double csq = T * (1.0 + 2.0 * diff_A_NkT * x + d2A_dx2 * x * x); // soundspeed squared
  if (csq > 0.0) {
    *c = sqrt(csq); // soundspeed
  } else {
    *c = 0.0;
  }
}


__kernel void k_lj_sph(const __global numtyp4 *restrict x_,
                       const __global numtyp4 *restrict v_,
                       const __global numtyp *restrict cv_,
                       const __global numtyp *restrict e_,
                       const __global numtyp *restrict rho_,
                       const __global numtyp *restrict de_,
                       const __global numtyp *restrict drho_,
                       const __global numtyp4 *restrict cuts,
                       const int lj_types,
                       const __global int * dev_nbor,
                       const __global int * dev_packed,
                       __global acctyp4 *restrict ans,
                       __global acctyp *restrict engv,
                       const int eflag, const int vflag, const int inum,
                       const int nbor_pitch,
                       const int t_per_atom, int domainDim){ //TODO: arguments?
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  double h, ih, ihsq, ihcub, wfd, fi, ci, fj, cj;
  double delVdotDelR, mu, fvisc, fpair, deltaE, imass, jmass;

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    double cvi; fetch(cvi, i, cv_tex);
    double ei; fetch(ei, i, cv_tex);
    double rhoi; fetch(rhoi, i, cv_tex);

    LJEOS2(rhoi,ei,cvi, &fi, &ci);
    fi /= (rhoi * rhoi);

    int itype=ix.w;
    imass = cuts[itype].z;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtype=jx.w;


      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;
      jmass = cuts[jtype].z;

      int mtype=itype*lj_types+jtype;
      if (rsq<cuts[mtype].z) {
        h = cuts[mtype].y;
        ih = 1.0/h;
        ihsq = ih * ih;
        ihcub = ihsq * ih;

        wfd = h - sqrt(rsq);

      if (domainDim == 3) {
          // Lucy Kernel, 3d
          // Note that wfd, the derivative of the weight function with respect to r,
          // is lacking a factor of r.
          // The missing factor of r is recovered by
          // (1) using delV . delX instead of delV . (delX/r) and
          // (2) using f[i][0] += delx * fpair instead of f[i][0] += (delx/r) * fpair
          wfd = -25.066903536973515383e0 * wfd * wfd * ihsq * ihsq * ihsq * ih;
        } else {
            // Lucy Kernel, 2d
            wfd = -19.098593171027440292e0 * wfd * wfd * ihsq * ihsq * ihsq;
        }

        double cvj; fetch(cvj, j, cv_tex);
        double ej; fetch(ej, j, cv_tex);
        double rhoj; fetch(rhoj, j, cv_tex);

        // function call to LJ EOS
        LJEOS2(rhoj, ej, cvj, &fj, &cj);
        fj /= (rhoj * rhoj);

        // apply long-range correction to model a LJ fluid with cutoff
        // this implies that the modelled LJ fluid has cutoff == SPH cutoff
        double lrc = - 11.1701 * (ihcub * ihcub * ihcub - 1.5 * ihcub);
        fi += lrc;
        fj += lrc;


        // dot product of velocity delta and distance vector
        delVdotDelR = delx * (iv.x - jv.x) + dely * (iv.y -jv.y)
            + delz * (iv.z - jv.z);

        // artificial viscosity (Monaghan 1992)
        if (delVdotDelR < 0.) {
          mu = h * delVdotDelR / (rsq + 0.01 * h * h);
          fvisc = -0.04 * (ci + cj) * mu / (rhoi + rhoj); //TODO: implement the viscosity
        } else {
          fvisc = 0.;
        }

        // total pair force & thermal energy increment
        fpair = -imass * jmass * (fi + fj + fvisc) * wfd;
        deltaE = -0.5 * fpair * delVdotDelR;

        f.x+=delx*fpair;
        f.y+=dely*fpair;
        f.z+=delz*fpair;

        double dei; fetch(dei, i, de_tex);
        double drhoi; fetch(drhoi, i, drho_tex);

        // and change in density
        drhoi += jmass * delVdotDelR * wfd;

        // change in thermal energy
        dei += deltaE;

        if (vflag>0) {
          virial[0] += delx*delx*fpair;
          virial[1] += dely*dely*fpair;
          virial[2] += delz*delz*fpair;
          virial[3] += delx*dely*fpair;
          virial[4] += delx*delz*fpair;
          virial[5] += dely*delz*fpair;
        }
      }

    } // for nbor TODO: write an adapted store_answer function
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}

__kernel void k_lj_sph_fast(const __global numtyp4 *restrict x_,
                       const __global numtyp4 *restrict v_,
                       const __global numtyp *restrict cv_,
                       const __global numtyp *restrict e_,
                       const __global numtyp *restrict rho_,
                       const __global numtyp *restrict de_,
                       const __global numtyp *restrict drho_,
                       const __global numtyp4 *restrict cuts_in,
                       const __global int * dev_nbor,
                       const __global int * dev_packed,
                       __global acctyp4 *restrict ans,
                       __global acctyp *restrict engv,
                       const int eflag, const int vflag, const int inum,
                       const int nbor_pitch,
                       const int t_per_atom, int domainDim){ //TODO: arguments?
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);
  double h, ih, ihsq, ihcub, wfd, fi, ci, fj, cj;
  double delVdotDelR, mu, fvisc, fpair, deltaE, imass, jmass;

  __local numtyp4 cuts[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    cuts[tid]=cuts_in[tid];
  }

  acctyp energy=(acctyp)0;
  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp virial[6];
  for (int i=0; i<6; i++)
    virial[i]=(acctyp)0;

  __syncthreads();

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    __local int n_stride;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    double cvi; fetch(cvi, i, cv_tex);
    double ei; fetch(ei, i, cv_tex);
    double rhoi; fetch(rhoi, i, cv_tex);

    LJEOS2(rhoi,ei,cvi, &fi, &ci);
    fi /= (rhoi * rhoi);
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);

    imass = cuts[itype].z;
    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtype=jx.w;


      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;
      jmass = cuts[jtype].z;

      int mtype=itype+jtype;
      if (rsq<cuts[mtype].z) {
        h = cuts[mtype].y;
        ih = 1.0/h;
        ihsq = ih * ih;
        ihcub = ihsq * ih;

        wfd = h - sqrt(rsq);

      if (domainDim == 3) {
          // Lucy Kernel, 3d
          // Note that wfd, the derivative of the weight function with respect to r,
          // is lacking a factor of r.
          // The missing factor of r is recovered by
          // (1) using delV . delX instead of delV . (delX/r) and
          // (2) using f[i][0] += delx * fpair instead of f[i][0] += (delx/r) * fpair
          wfd = -25.066903536973515383e0 * wfd * wfd * ihsq * ihsq * ihsq * ih;
        } else {
            // Lucy Kernel, 2d
            wfd = -19.098593171027440292e0 * wfd * wfd * ihsq * ihsq * ihsq;
        }

        double cvj; fetch(cvj, j, cv_tex);
        double ej; fetch(ej, j, cv_tex);
        double rhoj; fetch(rhoj, j, cv_tex);

        // function call to LJ EOS
        LJEOS2(rhoj, ej, cvj, &fj, &cj);
        fj /= (rhoj * rhoj);

        // apply long-range correction to model a LJ fluid with cutoff
        // this implies that the modelled LJ fluid has cutoff == SPH cutoff
        double lrc = - 11.1701 * (ihcub * ihcub * ihcub - 1.5 * ihcub);
        fi += lrc;
        fj += lrc;


        // dot product of velocity delta and distance vector
        delVdotDelR = delx * (iv.x - jv.x) + dely * (iv.y -jv.y)
            + delz * (iv.z - jv.z);

        // artificial viscosity (Monaghan 1992)
        if (delVdotDelR < 0.) {
          mu = h * delVdotDelR / (rsq + 0.01 * h * h);
          fvisc = -0.04 * (ci + cj) * mu / (rhoi + rhoj); //TODO: implement the viscosity
        } else {
          fvisc = 0.;
        }

        // total pair force & thermal energy increment
        fpair = -imass * jmass * (fi + fj + fvisc) * wfd;
        deltaE = -0.5 * fpair * delVdotDelR;

        f.x+=delx*fpair;
        f.y+=dely*fpair;
        f.z+=delz*fpair;

        double dei; //fetch(dei, i, dcv_tex);
        double drhoi; //fetch(drhoi, i, dcv_tex);

        // and change in density
        //drho_[i] += jmass * delVdotDelR * wfd;

        // change in thermal energy
        //de_[i] += deltaE;

        if (vflag>0) {
          virial[0] += delx*delx*fpair;
          virial[1] += dely*dely*fpair;
          virial[2] += delz*delz*fpair;
          virial[3] += delx*dely*fpair;
          virial[4] += delx*delz*fpair;
          virial[5] += dely*delz*fpair;
        }
      }

    } // for nbor TODO: write an adapted store_answer function
    store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                  ans,engv);
  } // if ii
}
